#include "hip/hip_runtime.h"
#include <string>
#include <cstring>
#include <iostream>

#include <opencv2/opencv.hpp>
#include "opencv2/core.hpp"
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/cudaimgproc.hpp"
#include "opencv2/imgcodecs.hpp"

#define BLOCK_SIZE (64)

void bubbleSort(uchar* buffer, size_t bufferSize)
{
    
    for(size_t i = 0; i <bufferSize; ++i){
        for(size_t j = i + 1; j < bufferSize; ++j){
            if(buffer[i] > buffer[j]){
                uchar tmp = buffer[i];
                buffer[i] = buffer[j];
                buffer[j] = tmp; 
            }
        }
    }
}

__global__ void medianFilter(unsigned char* inputImage, unsigned char* outputImage, int imageWidth, int imageHeight, size_t channels, int windowWidth, int windowsHeight)
{
    
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    if( (x < (windowWidth/2)) || (y < (windowsHeight / 2)) || (x >= (imageWidth - (windowWidth/2))) || (y >= (imageHeight - (windowsHeight / 2))) ){
        for(size_t ch = 0; ch < 3; ++ch)
        {
            outputImage[y * imageWidth * channels + x * channels + ch] = inputImage[y * imageWidth * channels + x * channels + ch];
        }
    }
    else{
        uchar *window = new uchar[windowWidth * windowsHeight];
        for(size_t ch = 0; ch < channels; ++ch){
            size_t iterator = 0;
            for(size_t row = (y - (windowsHeight / 2)); row <= (y + (windowsHeight / 2)); ++row){
                for(size_t col = (x - (windowWidth / 2)); col <= (x + (windowWidth / 2)); ++col){
                    window[iterator] = inputImage[row * imageWidth * channels + col * channels + ch];
                    ++iterator;
                }
            }
            // bubble sort
            for(size_t i = 0; i < windowWidth * windowsHeight; ++i){
                for(size_t j = i + 1; j < windowWidth * windowsHeight; ++j){
                    if( window[i] > window[j] ){
                        uchar tmp = window[i];
                        window[i] = window[j];
                        window[j] = tmp; 
                    }
                }
            }
            outputImage[y * imageWidth * channels + x * channels + ch] = window[(windowWidth * windowsHeight) / 2];
        }
        delete window;
    }
}

__global__ void medianFilterShared(unsigned char* inputImage, unsigned char* outputImage, int imageWidth, int imageHeight, int channels, int windowWidth, int windowsHeight)
{
    
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ unsigned char sharedmem[BLOCK_SIZE + 2][BLOCK_SIZE + 2][3];

    bool is_x_left = ( 0 == threadIdx.x);
    bool is_x_right = ( (BLOCK_SIZE - 1) == threadIdx.x);
    bool is_y_top = ( 0 == threadIdx.y);
    bool is_y_bottom = ( (BLOCK_SIZE - 1 ) == threadIdx.y);

    if(is_x_left){
        if(x > 0)
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x][threadIdx.y + 1][ch] = inputImage[y * imageWidth * channels + (x - 1) * channels + ch];
            }
        }
        else{
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x][threadIdx.y + 1][ch] = 0;
            }
        }
    }
    else if(is_x_right){
        if(x < (imageWidth - 1))
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x + 2][threadIdx.y + 1][ch] = inputImage[y * imageWidth * channels + (x + 1) * channels + ch];;
            }

        }
        else
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x + 2][threadIdx.y + 1][ch] = 0;
            }
        }
    }
    if (is_y_top){
        if(y > 0)
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x + 1][threadIdx.y][ch] = inputImage[(y - 1) * imageWidth * channels + x * channels + ch];
            }
            if(is_x_left)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x][threadIdx.y][ch] = inputImage[(y - 1) * imageWidth * channels + (x - 1) * channels + ch];    
                    
                }
            }
            else if(is_x_right)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x + 2][threadIdx.y][ch] = inputImage[(y - 1) * imageWidth * channels + (x + 1) * channels + ch];;
                }   
            }
        }
        else
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x + 1][threadIdx.y][ch] = 0;
            }
            if(is_x_left)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x][threadIdx.y][ch] = 0;
                }
            }
            else if(is_x_right)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x + 2][threadIdx.y][ch] = 0;
                }   
            }
        }
	}
	else if (is_y_bottom){
        if(y < (imageHeight - 1))
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x + 1][threadIdx.y + 2][ch] = inputImage[(y + 1) * imageWidth * channels + x * channels + ch];
            }
            if(is_x_left)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x][threadIdx.y + 2][ch] = inputImage[(y + 1) * imageWidth * channels + (x - 1)* channels + ch];
                }
            }
            else if(is_x_right)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x + 2][threadIdx.y + 2][ch] = inputImage[(y + 1) * imageWidth * channels + (x + 1)* channels + ch];;
                }
            }
        }
        else
        {
            for(size_t ch = 0; ch < channels; ++ch)
            {
                sharedmem[threadIdx.x + 1][threadIdx.y + 2][ch] = 0;
            }
            if(is_x_left)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x][threadIdx.y + 2][ch] = 0;
                }
            }
            else if(is_x_right)
            {
                for(size_t ch = 0; ch < channels; ++ch)
                {
                    sharedmem[threadIdx.x + 2][threadIdx.y + 2][ch] = 0;
                }
            }
        }
	}

    for(size_t ch = 0; ch < channels; ++ch)
    {
        sharedmem[threadIdx.x + 1][threadIdx.y + 1][ch] = inputImage[y * imageWidth * channels + x * channels + ch];
    }
    
    __syncthreads();

    uchar *window = new uchar[windowWidth * windowsHeight];

    for(size_t ch = 0; ch < channels; ++ch)
    {
        size_t iterator = 0;
        for (size_t col = threadIdx.x; col <= threadIdx.x + 2; ++col)
        {
            for(size_t row = threadIdx.y; row <= threadIdx.y + 2; ++row)
            {
                window[iterator] = sharedmem[col][row][ch]; 
                ++iterator;
            }
        }
        // bubble sort
        for(size_t i = 0; i < windowWidth * windowsHeight; ++i){
            for(size_t j = i + 1; j < windowWidth * windowsHeight; ++j){
                if( window[i] > window[j] ){
                    uchar tmp = window[i];
                    window[i] = window[j];
                    window[j] = tmp; 
                }
            }
        }
        outputImage[y * imageWidth * channels + x * channels + ch] = window[(windowWidth * windowsHeight) / 2];
    }

    delete window;
    __syncthreads();
}

int main(int argc, char** argv)
{
    if(argc >= 4)
    {
        cv::Mat img;
        img = cv::imread(argv[3]);

        if(img.empty())
        {
            std::cout << "Could not open or find the image" << std::endl;
            return -1;
        }

        unsigned int window_rows{(unsigned int) std::stoul(argv[1])};
        unsigned int window_cols{(unsigned int) std::stoul(argv[2])};

        int width = img.cols;
        int height = img.rows;
        int channels = img.channels();

        hipError_t status;
        hipEvent_t start, stop;
        float time;

        uint64_t imageSize = width * height * channels * sizeof(uchar);

        unsigned char *outputImageHost = (unsigned char *) malloc(imageSize);
        unsigned char *inputImageDevice;
        unsigned char *outputImageDevice;

        hipMalloc<unsigned char>(&inputImageDevice, imageSize);
        status = hipGetLastError();
        if (status != hipSuccess) {                     
            std::cout << "Kernel failed for hipMalloc : " << hipGetErrorString(status) << std::endl;
            return -1;
        }

        hipMalloc<unsigned char>(&outputImageDevice, imageSize);
        status = hipGetLastError();
        if (status != hipSuccess) {                     
            std::cout << "Kernel failed for hipMalloc : " << hipGetErrorString(status) << std::endl;
            return -1;
        }

        hipMemcpy(inputImageDevice, img.ptr(), imageSize, hipMemcpyHostToDevice);
        status = hipGetLastError();
        if (status != hipSuccess) {                     
            std::cout << "Kernel failed for hipMemcpy hipMemcpyHostToDevice: " << hipGetErrorString(status) << std::endl;
            hipFree(inputImageDevice);
            return -1;
        }
        // hipMalloc((void**)&outputImageDevice, imageSize);
        // const dim3 grid (((width % BLOCK_WIDTH) != 0) ? (width / BLOCK_WIDTH + 1) : (width / BLOCK_WIDTH), ((height % BLOCK_HEIGHT) != 0) ? (height / BLOCK_HEIGHT + 1) : (height / BLOCK_HEIGHT), 1);
        // const dim3 block ((int)ceil((float)width / (float)BLOCK_WIDTH), (int)ceil((float)height / (float)BLOCK_HEIGHT));

        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);
        const dim3 grid(cv::cuda::device::divUp(width, block.x), cv::cuda::device::divUp(height, block.y));

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        medianFilter<<<grid, block>>>(inputImageDevice, outputImageDevice, width, height, channels, window_cols,  window_rows);
        status = hipGetLastError(); 

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        
        std::cout << "GPU time in milliseconds: " << time << " ms" << std::endl;

        hipMemcpy(inputImageDevice, img.ptr(), imageSize, hipMemcpyHostToDevice);
        status = hipGetLastError();
        if (status != hipSuccess) {                     
            std::cout << "Kernel function failed: " << hipGetErrorString(status) << std::endl;
            hipFree(inputImageDevice);
            return -1;
        }

        
        if(3 == window_cols && 3 == window_rows && channels <= 3)
        {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
    
            medianFilterShared<<<grid, block>>>(inputImageDevice, outputImageDevice, width, height, channels, window_cols,  window_rows);
            status = hipGetLastError(); 
    
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&time, start, stop);
    
            hipMemcpy(inputImageDevice, img.ptr(), imageSize, hipMemcpyHostToDevice);
            status = hipGetLastError();
            if (status != hipSuccess) {                     
                std::cout << "Shared memory kernel function failed: " << hipGetErrorString(status) << std::endl;
                hipFree(inputImageDevice);
                return -1;
            }
    
            std::cout << "GPU time for shared memory in milliseconds: " << time << " ms" << std::endl;
            
            hipMemcpy(outputImageHost, outputImageDevice, imageSize, hipMemcpyDeviceToHost);
            status = hipGetLastError();
            if (status != hipSuccess) {                     
                std::cout << "Kernel failed for hipMemcpy hipMemcpyHostToDevice: " << hipGetErrorString(status) << std::endl;
                hipFree(inputImageDevice);
                return -1;
            }
        }

        cv::Mat output_image = cv::Mat(height, width, img.type(), outputImageHost);

        if(5 == argc)
        {
            cv::imwrite(argv[4], output_image);
        }
        else
        {
            cv::imwrite("output.png", output_image);
        }

        hipFree(inputImageDevice);
        hipFree(outputImageDevice);
        free(outputImageHost);

        return 0;
    }
    else
    {
        std::cout << "usage: median_filter_normal wh ww input_image <output_directory>" << std::endl << std::endl;
        std::cout << "Applies median filter to the image" << std::endl << std::endl;
        std::cout << "Positional arguments:" << std::endl;
        std::cout << std::left << std::setw(30) << "  wh" << "kernel height, odd number" << std::endl;
        std::cout << std::left << std::setw(30) << "  ww" << "kernel width, odd number" << std::endl;
        std::cout << std::left << std::setw(30) << "  input_image" << "input image directory" << std::endl;
        std::cout << std::left << std::setw(30) << "  <output_directory>" << "optional output directory, if not present then image is saved to output.png" << std::endl;
        return -1;
    }
}